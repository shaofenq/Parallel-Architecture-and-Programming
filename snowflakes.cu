#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}
__global__ void
kernel1(int* device_data, int N, int twod, int twod1) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index * twod1;
    if ((i+twod1-1) < N && (i+twod-1) < N) {
        device_data[i+twod1-1] += device_data[i+twod-1];
        // printf("kernel1: %d %d %d %d %d\n", index, twod, twod1, device_data[i+twod1-1], device_data[i+twod-1]);
    }
}

__global__ void
kernel2(int* device_data, int N, int twod, int twod1) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("kernel2 twod=%d twod1=%d \n", twod, twod1);
    int i = index * twod1;

    if (i+twod1-1<N && i+twod-1<N) {
        int t = device_data[i+twod-1];
        device_data[i+twod-1] = device_data[i+twod1-1];
        device_data[i+twod1-1] += t;
    }
    
    // if (index < N) {
    //     result[index] = alpha * x[index] + y[index];
    // }
}
__global__ void
setLast(int* device_data, int N) {
    device_data[N-1]=0;
}

void exclusive_scan(int* device_data, int N)
{
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */
    const int threadsPerBlock = 512;
    // const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    int N1 = nextPow2(N);
    printf("N1=%d\n",N1);
    // printf("blocks=%d \n",blocks);
    for (int twod=1; twod < N1; twod *=2) {
        int twod1 = twod*2;
        //parallel for
        int blocks = (N1/twod1+threadsPerBlock-1)/threadsPerBlock;
        printf("blocks=%d \n",blocks);
        kernel1<<<blocks, threadsPerBlock>>>(device_data, N1, twod, twod1);
    }
    //device_data[N-1] = 0;
    setLast<<<1,1>>>(device_data, N1);
    // printf("step2\n");
    for (int twod=N1/2; twod >= 1; twod /= 2) {
        int twod1 = twod*2;
        //parallel for 
        int blocks = (N1/twod1+threadsPerBlock-1)/threadsPerBlock;
        kernel2<<<blocks, threadsPerBlock>>>(device_data, N1, twod, twod1);
    }
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    printf("inarray:");
    for (int i=0; i < 10; i++) {
        printf("%d ", inarray[i]);
    }
    printf("\n");
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    printf("end - inarray %ld \n", end - inarray);
    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void
kernel3(int* device_output, int* device_input, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > 0 && index < N-1 && device_input[index] > device_input[index-1] && 
        device_input[index] > device_input[index+1]) {
        device_output[index] = 1;
    }
}

__global__ void
kernel4(int* device_output, int* device_input, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > 0 && device_input[index] > device_input[index-1]) {
        // printf("index: %d %d \n",device_input[index]-1, index);
        device_output[device_input[index]-1] = index-1;
    }
}



int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    const int threadsPerBlock = 512;
    int blocks = (length+threadsPerBlock-1)/threadsPerBlock;
    kernel3<<<blocks, threadsPerBlock>>>(device_output, device_input, length);
    printf("step1\n");

    exclusive_scan(device_output, length);
    printf("step2\n");
    
    hipMemcpy(device_input, device_output, length * sizeof(int), hipMemcpyHostToHost);
    hipMemset(device_output, 0, length * sizeof(int));

    kernel4<<<blocks, threadsPerBlock>>>(device_output, device_input, length);
    printf("step3\n");
    // int count = device_input[length-1];
    int count = 0;
    hipMemcpy(&count, device_input+length-1, sizeof(int), hipMemcpyDeviceToHost);
    return count;
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
